#include "hip/hip_runtime.h"
#include "DepthAdaptiveSuperpixel.h"

__global__ void init_LD(
	DepthAdaptiveSuperpixel::label_distance* pixel_ld,
	int width,
	int height,
	int2 cluster_num,
	int2 window_size){
		int x = blockIdx.x*blockDim.x+threadIdx.x;
		int y = blockIdx.y*blockDim.y+threadIdx.y;
		//current label
		pixel_ld[y*width+x].l = (y/window_size.y)*cluster_num.x + (x/window_size.x);
		pixel_ld[y*width+x].d = 999999.9f;
}

template<int blockSize>
__global__ void sampleInitialClusters(
	cv::gpu::GpuMat color_input,
	float3*		input_points,
	DepthAdaptiveSuperpixel::superpixel*	mean,
	float3*		sp_centers,
	int width,
	int height,
	int2 window_size){
		__shared__ float gradient[blockSize];
		__shared__ int	 arounds_x[blockSize];
		__shared__ int	 arounds_y[blockSize];
		//center of cluster
		int2 center;
		center.x = blockIdx.x*window_size.x + window_size.x/2;
		center.y = blockIdx.y*window_size.y + window_size.y/2;
		//around center
		int2 around;
		around.x = center.x + threadIdx.x - blockDim.x/2;
		around.y = center.y + threadIdx.y - blockDim.y/2;
		//thread id
		int tid = threadIdx.y*blockDim.x+threadIdx.x;
		//compute gradient
		float sumG = 0.0f;
		int count = 0;
		float g;
		for(int yy = -5; yy <= 5; yy++){
			for(int xx = -5; xx <= 5; xx++){
				int lx = xx + around.x;
				lx = lx > 0 ? lx : 0;
				lx = lx < width ? lx : width - 1;

				int ly = yy + around.y;
				ly = ly > 0 ? ly : 0;
				ly = ly < height ? ly : height - 1;

				g = sqrtf(pow((float)color_input.data[(around.y*width+around.x)*3]-(float)color_input.data[(yy*width+xx)*3], 2) +
					pow((float)color_input.data[(around.y*width+around.x)*3+1]-(float)color_input.data[(yy*width+xx)*3+1], 2) +
					pow((float)color_input.data[(around.y*width+around.x)*3+2]-(float)color_input.data[(yy*width+xx)*3+2], 2));
				count += g > 0.0 ? 1 : 0;
				sumG += g;
			}
		}
		//�S�_�ɂ���gradient���v�Z
		gradient[tid] = sumG/(float)count;
		arounds_x[tid] = around.x;
		arounds_y[tid] = around.y;
		__syncthreads();

		//compute min value
		if(blockSize >= 1024){
			if(tid < 512){
				if(gradient[tid] > gradient[tid+512]){
					gradient[tid] = gradient[tid+512];
					arounds_x[tid] = arounds_x[tid+512];
					arounds_y[tid] = arounds_y[tid+512];
					__syncthreads();
				}
			}
		}
		if(blockSize >= 512){
			if(tid < 256){
				if(gradient[tid] > gradient[tid+256]){
					gradient[tid] = gradient[tid+256];
					arounds_x[tid] = arounds_x[tid+256];
					arounds_y[tid] = arounds_y[tid+256];
					__syncthreads();
				}
			}
		}
		if(blockSize >= 256){
			if(tid < 128){
				if(gradient[tid] > gradient[tid+128]){
					gradient[tid] = gradient[tid+128];
					arounds_x[tid] = arounds_x[tid+128];
					arounds_y[tid] = arounds_y[tid+128];
					__syncthreads();
				}
			}
		}
		if(blockSize >= 128){
			if(tid < 64){
				if(gradient[tid] > gradient[tid+64]){
					gradient[tid] = gradient[tid+64];
					arounds_x[tid] = arounds_x[tid+64];
					arounds_y[tid] = arounds_y[tid+64];
					__syncthreads();
				}
			}
		}
		if(tid < 32){
			if(blockSize >= 64){
				if(gradient[tid] > gradient[tid+32]){
					gradient[tid] = gradient[tid+32];
					arounds_x[tid] = arounds_x[tid+32];
					arounds_y[tid] = arounds_y[tid+32];
				}
			}
			if(blockSize >= 32){
				if(gradient[tid] > gradient[tid+16]){
					gradient[tid] = gradient[tid+16];
					arounds_x[tid] = arounds_x[tid+16];
					arounds_y[tid] = arounds_y[tid+16];
				}
			}
			if(blockSize >= 16){
				if(gradient[tid] > gradient[tid+8]){
					gradient[tid] = gradient[tid+8];
					arounds_x[tid] = arounds_x[tid+8];
					arounds_y[tid] = arounds_y[tid+8];
				}
			}
			if(blockSize >= 8){
				if(gradient[tid] > gradient[tid+4]){
					gradient[tid] = gradient[tid+4];
					arounds_x[tid] = arounds_x[tid+4];
					arounds_y[tid] = arounds_y[tid+4];
				}
			}
			if(blockSize >= 4){
				if(gradient[tid] > gradient[tid+2]){
					gradient[tid] = gradient[tid+2];
					arounds_x[tid] = arounds_x[tid+2];
					arounds_y[tid] = arounds_y[tid+2];
				}
			}
			if(blockSize >= 2){
				if(gradient[tid] > gradient[tid+1]){
					gradient[tid] = gradient[tid+1];
					arounds_x[tid] = arounds_x[tid+1];
					arounds_y[tid] = arounds_y[tid+1];
				}
			}
		}
		//store center point
		if(tid == 0){
			int2 smooth;
			smooth.x = arounds_x[0];
			smooth.y = arounds_y[0];
			mean[blockIdx.y*(width/window_size.x)+blockIdx.x].x = (int)smooth.x;
			mean[blockIdx.y*(width/window_size.x)+blockIdx.x].y = (int)smooth.y;
			mean[blockIdx.y*(width/window_size.x)+blockIdx.x].r = color_input.data[(smooth.y*width+smooth.x)*3];
			mean[blockIdx.y*(width/window_size.x)+blockIdx.x].g = color_input.data[(smooth.y*width+smooth.x)*3+1];
			mean[blockIdx.y*(width/window_size.x)+blockIdx.x].b = color_input.data[(smooth.y*width+smooth.x)*3]+2;
			sp_centers[blockIdx.y*(width/window_size.x)+blockIdx.x].x = input_points[smooth.y*width+smooth.x].x;
			sp_centers[blockIdx.y*(width/window_size.x)+blockIdx.x].y = input_points[smooth.y*width+smooth.x].y;
			sp_centers[blockIdx.y*(width/window_size.x)+blockIdx.x].z = input_points[smooth.y*width+smooth.x].z;
		}

}

template<int blockSize>
__global__ void calculateLD(
	cv::gpu::GpuMat color_input,
	float3*		input_points,
	DepthAdaptiveSuperpixel::label_distance* pixel_ld,
	DepthAdaptiveSuperpixel::superpixel* mean,
	float3*		sp_centers,
	int* labels,
	int2 window_size,
	int width,
	int height,
	float color_sigma,
	float spatial_sigma,
	float depth_sigma,
	int2 cluster_num,
	cv::gpu::GpuMat intr){
		__shared__ int label_shared[blockSize];
		__shared__ float distance_shared[blockSize];
		//thread id
		int tid = threadIdx.y*blockDim.x+threadIdx.x;	
		//current pixel
		int x = blockIdx.x;
		int y = blockIdx.y;
		//current cluster pos
		int2 current_cluster;
		current_cluster.x = pixel_ld[y*width+x].l%cluster_num.x;
		current_cluster.y = pixel_ld[y*width+x].l/cluster_num.x;
		//assign threads around each cluster
		int2 ref_cluster;
		ref_cluster.x = current_cluster.x-blockDim.x/2+threadIdx.x;
		ref_cluster.y = current_cluster.y-blockDim.y/2+threadIdx.y;
		//ref cluster id
		int ref_cluster_id = ref_cluster.y*cluster_num.x+ref_cluster.x;
		if(ref_cluster.x>=0 && ref_cluster.x<cluster_num.x && 
			ref_cluster.y>=0 && ref_cluster.y<cluster_num.y){
				int2 ref_center;
				ref_center.x = (int)mean[ref_cluster_id].x;
				ref_center.y = (int)mean[ref_cluster_id].y;
				//calculate distance
				float color_distance = pow((float)color_input.data[(y*width+x)*3]-(float)mean[ref_cluster_id].r, 2) +
											pow((float)color_input.data[(y*width+x)*3+1]-(float)mean[ref_cluster_id].g, 2) +
												pow((float)color_input.data[(y*width+x)*3+2]-(float)mean[ref_cluster_id].b, 2);
				////calculate pixel distance
				////float spatial_distance = sqrtf(pow((float)(x-ref_center.x), 2) + pow((float)(y-ref_center.y), 2));
				//float spatial_distance;
				//if(input_points[y*width+x].z > 50.0f &&  sp_centers[ref_cluster_id].z > 50.0f){
				//	float distance_3d = pow(input_points[y*width+x].x-sp_centers[ref_cluster_id].x, 2.0f) +
				//							pow(input_points[y*width+x].y-sp_centers[ref_cluster_id].y, 2.0f) +
				//								pow(input_points[y*width+x].z-sp_centers[ref_cluster_id].z, 2.0f);
				//	float sp_size = pow((float)(window_size.x+window_size.y)/2.0f, 2.0f);
				//	float focal = (intr.data[0]+intr.data[4])/2.0f;
				//	//spatial_distance = distance_3d*sp_size*pow(sqrt(distance_3d)/focal, 2.0f);
				//	//spatial_distance = distance_3d*sp_size/**pow((intr.data[0]+intr.data[4])/(2.0f*input_points[y*width+x].z), 2.0f)*/;
				//	
				//	//spatial_distance = (pow((float)(x-mean[ref_cluster_id].x), 2) + pow((float)(y-mean[ref_cluster_id].y), 2) + pow((input_points[y*width+x].z-sp_centers[ref_cluster_id].z)/input_points[y*width+x].z, 2.0f)) * 
				//	//							pow((float)(window_size.x+window_size.y)/2.0f, 2);
				//	//spatial_distance = (pow((float)(x-mean[ref_cluster_id].x), 2) + pow((float)(y-mean[ref_cluster_id].y), 2) +  pow(input_points[y*width+x].z-sp_centers[ref_cluster_id].z, 2.0f)) * 
				//	//							 pow((float)(window_size.x+window_size.y)/2.0f, 2);
				//
				//}
				//else
				//	spatial_distance = (pow((float)(x-mean[ref_cluster_id].x), 2) + pow((float)(y-mean[ref_cluster_id].y), 2)) * 
				//								pow((float)(window_size.x+window_size.y)/2.0f, 2);
				float spatial_distance = sqrtf(pow((float)(x-mean[ref_cluster_id].x), 2) + pow((float)(y-mean[ref_cluster_id].y), 2)) * 
																						pow((float)(window_size.x+window_size.y)/2.0f, 2);
				float depth_distance = 0.0f;
				if(input_points[y*width+x].z > 50.0f &&  sp_centers[ref_cluster_id].z > 50.0f){
					float diff = abs(input_points[y*width+x].z - sp_centers[ref_cluster_id].z );
					//if( diff > 100.0f){
					//spatial_distance += abs(input_points[y*width+x].z - sp_centers[ref_cluster_id].z);
					float focal = (intr.data[0]+intr.data[4])/2.0f;
					//spatial_distance += (diff/input_points[y*width+x].z)*focal*pow((float)(window_size.x+window_size.y)/2.0f, 2.0f);
					//depth_distance = (diff/input_points[y*width+x].z)*focal*pow((float)(window_size.x+window_size.y)/2.0f, 2.0f);
					depth_distance = diff;
					//}
				}
				float sum_sigma = spatial_sigma+color_sigma+depth_sigma;
				//set current ld
				distance_shared[tid] = color_distance*pow(color_sigma/sum_sigma, 2.0f) + spatial_distance*pow(spatial_sigma/sum_sigma, 2.0f) + depth_distance*pow(depth_sigma/sum_sigma, 2.0f);
				label_shared[tid] = ref_cluster.y*cluster_num.x+ref_cluster.x;		
		}
		else{
			distance_shared[tid] = pixel_ld[y*width+x].d;
			label_shared[tid] = pixel_ld[y*width+x].l;
		}
		__syncthreads();

		//assign cluster label
		if(blockSize >= 1024){
			if(tid < 512){
				if(distance_shared[tid] > distance_shared[tid+512]){
					label_shared[tid] = label_shared[tid+512];
					distance_shared[tid] = distance_shared[tid+512];
				}
				__syncthreads();
			}
		}
		if(blockSize >= 512){
			if(tid < 256){
				if(distance_shared[tid]> distance_shared[tid+256]){
					label_shared[tid] = label_shared[tid+256];
					distance_shared[tid] = distance_shared[tid+256];
				}
				__syncthreads();
			}
		}
		if(blockSize >= 256){
			if(tid < 128){
				if(distance_shared[tid] > distance_shared[tid+128]){
					label_shared[tid] = label_shared[tid+128];
					distance_shared[tid] = distance_shared[tid+128];
				}
				__syncthreads();
			}
		}
		if(blockSize >= 128){
			if(tid < 64){
				if(distance_shared[tid] > distance_shared[tid+64]){
					label_shared[tid] = label_shared[tid+64];
					distance_shared[tid] = distance_shared[tid+64];
				}
				__syncthreads();
			}
		}
		if(tid < 32){
			if(blockSize >= 64){
				if(distance_shared[tid] > distance_shared[tid+32]){
					label_shared[tid] = label_shared[tid+32];
					distance_shared[tid] = distance_shared[tid+32];
				}
			}
			if(blockSize >= 32){
				if(distance_shared[tid] > distance_shared[tid+16]){
					label_shared[tid] = label_shared[tid+16];
					distance_shared[tid] = distance_shared[tid+16];
				}
			}
			if(blockSize >= 16){
				if(distance_shared[tid] > distance_shared[tid+8]){
					label_shared[tid] = label_shared[tid+8];
					distance_shared[tid] = distance_shared[tid+8];
				}
			}
			if(blockSize >= 8){
				if(distance_shared[tid] > distance_shared[tid+4]){
					label_shared[tid] = label_shared[tid+4];
					distance_shared[tid] = distance_shared[tid+4];
				}
			}
			if(blockSize >= 4){
				if(distance_shared[tid] > distance_shared[tid+2]){
					label_shared[tid] = label_shared[tid+2];
					distance_shared[tid] = distance_shared[tid+2];
				}
			}
			if(blockSize >= 2){
				if(distance_shared[tid] > distance_shared[tid+1]){
					label_shared[tid] = label_shared[tid+1];
					distance_shared[tid] = distance_shared[tid+1];
				}
			}
		}
		//store center point
		if(tid == 0){
			pixel_ld[y*width+x].l = label_shared[0];
			pixel_ld[y*width+x].d = distance_shared[0];
			labels[y*width+x] = label_shared[0];
		}
		if(input_points[y*width+x].z < 50.0f && depth_sigma != 0.0f){
			pixel_ld[y*width+x].l = -1;
			pixel_ld[y*width+x].d = 0.0f;
			labels[y*width+x] = -1;
		}
}

template<int blockSize>
__global__ void analyzeClusters(
	cv::gpu::GpuMat color_input,
	float3*		input_points,
	DepthAdaptiveSuperpixel::label_distance* pixel_ld,
	DepthAdaptiveSuperpixel::superpixel* mean,
	float3*		sp_centers,
	int2 window_size,
	int2 cluster_num,
	int width,
	int height,
	cv::gpu::GpuMat intr){
		//4*10=40 Byte 16384/40 = 20*20 threads
		__shared__ int r_shared[blockSize];
		__shared__ int g_shared[blockSize];
		__shared__ int b_shared[blockSize];
		__shared__ int x_shared[blockSize];
		__shared__ int y_shared[blockSize];
		__shared__ int size_shared[blockSize];
		//3d info
		__shared__ float xw_shared[blockSize];
		__shared__ float yw_shared[blockSize];
		__shared__ float zw_shared[blockSize];
		__shared__ int num_of_points[blockSize];
		//thread id
		int tid = threadIdx.y*blockDim.x+threadIdx.x;
		r_shared[tid] = 0;
		g_shared[tid] = 0;
		b_shared[tid] = 0;
		x_shared[tid] = 0;
		y_shared[tid] = 0;
		size_shared[tid] = 0;
		//3d info
		xw_shared[tid] = 0.0f;
		yw_shared[tid] = 0.0f;
		zw_shared[tid] = 0.0f;
		num_of_points[tid] = 0;
		//current cluster
		int2 cluster_pos;
		cluster_pos.x = blockIdx.x;
		cluster_pos.y = blockIdx.y;
		int cluster_id = cluster_pos.y*cluster_num.x+cluster_pos.x;
		//assign threads around cluster
		int2 arounds;
		int2 ref_pixels;
		ref_pixels.x = window_size.x*2/blockDim.x+1;
		ref_pixels.y = window_size.y*2/blockDim.y+1;
		for(int yy=0; yy<ref_pixels.y; yy++){
			for(int xx=0; xx<ref_pixels.x; xx++){
				arounds.x = mean[cluster_id].x+(threadIdx.x-blockDim.x/2)*ref_pixels.x+xx;
				arounds.y = mean[cluster_id].y+(threadIdx.y-blockDim.y/2)*ref_pixels.y+yy;
				if(arounds.x>=0 && arounds.x<width && arounds.y>=0 && arounds.y<height){
					int around_id = pixel_ld[arounds.y*width+arounds.x].l;
					if(around_id == cluster_id){
						int r = (int)color_input.data[(arounds.y*width+arounds.x)*3];
						int g = (int)color_input.data[(arounds.y*width+arounds.x)*3+1];
						int b = (int)color_input.data[(arounds.y*width+arounds.x)*3+2];
						r = r>255 ? 255:r;
						g = g>255 ? 255:g;
						b = b>255 ? 255:b;
						r = r<0 ? 0:r;
						g = g<0 ? 0:g;
						b = b<0 ? 0:b;
						r_shared[tid] += r;
						g_shared[tid] += g;
						b_shared[tid] += b;
						x_shared[tid] += arounds.x;
						y_shared[tid] += arounds.y;
						size_shared[tid] += 1;
						xw_shared[tid] += input_points[arounds.y*width+arounds.x].x;
						yw_shared[tid] += input_points[arounds.y*width+arounds.x].y;
						zw_shared[tid] += input_points[arounds.y*width+arounds.x].z;
						num_of_points[tid] += input_points[arounds.y*width+arounds.x].z>50.0f ? 1:0;
					}
				}
			}
		}

		__syncthreads();
		//calculate average
		if(blockSize >= 1024){
			if(tid < 512){
				r_shared[tid] += r_shared[tid+512];
				g_shared[tid] += g_shared[tid+512];
				b_shared[tid] += b_shared[tid+512];
				x_shared[tid] += x_shared[tid+512];
				y_shared[tid] += y_shared[tid+512];
				xw_shared[tid] += xw_shared[tid+512];
				yw_shared[tid] += yw_shared[tid+512];
				zw_shared[tid] += zw_shared[tid+512];
				size_shared[tid] += size_shared[tid+512];
				num_of_points[tid] += num_of_points[tid+512];
				__syncthreads();
			}
		}
		if(blockSize >= 512){
			if(tid < 256){
				r_shared[tid] += r_shared[tid+256];
				g_shared[tid] += g_shared[tid+256];
				b_shared[tid] += b_shared[tid+256];
				x_shared[tid] += x_shared[tid+256];
				y_shared[tid] += y_shared[tid+256];
				xw_shared[tid] += xw_shared[tid+256];
				yw_shared[tid] += yw_shared[tid+256];
				zw_shared[tid] += zw_shared[tid+256];
				size_shared[tid] += size_shared[tid+256];
				num_of_points[tid] += num_of_points[tid+256];
				__syncthreads();
			}
		}
		if(blockSize >= 256){
			if(tid < 128){
				r_shared[tid] += r_shared[tid+128];
				g_shared[tid] += g_shared[tid+128];
				b_shared[tid] += b_shared[tid+128];
				x_shared[tid] += x_shared[tid+128];
				y_shared[tid] += y_shared[tid+128];
				xw_shared[tid] += xw_shared[tid+128];
				yw_shared[tid] += yw_shared[tid+128];
				zw_shared[tid] += zw_shared[tid+128];
				size_shared[tid] += size_shared[tid+128];
				num_of_points[tid] += num_of_points[tid+128];
				__syncthreads();
			}
		}
		if(blockSize >= 128){
			if(tid < 64){
				r_shared[tid] += r_shared[tid+64];
				g_shared[tid] += g_shared[tid+64];
				b_shared[tid] += b_shared[tid+64];
				x_shared[tid] += x_shared[tid+64];
				y_shared[tid] += y_shared[tid+64];
				xw_shared[tid] += xw_shared[tid+64];
				yw_shared[tid] += yw_shared[tid+64];
				zw_shared[tid] += zw_shared[tid+64];
				size_shared[tid] += size_shared[tid+64];
				num_of_points[tid] += num_of_points[tid+64];
				__syncthreads();
			}
		}
		if(tid < 32){
			if(blockSize >= 64){
				r_shared[tid] += r_shared[tid+32];
				g_shared[tid] += g_shared[tid+32];
				b_shared[tid] += b_shared[tid+32];
				x_shared[tid] += x_shared[tid+32];
				y_shared[tid] += y_shared[tid+32];
				xw_shared[tid] += xw_shared[tid+32];
				yw_shared[tid] += yw_shared[tid+32];
				zw_shared[tid] += zw_shared[tid+32];
				size_shared[tid] += size_shared[tid+32];
				num_of_points[tid] += num_of_points[tid+32];
			}
			if(blockSize >= 32){
				r_shared[tid] += r_shared[tid+16];
				g_shared[tid] += g_shared[tid+16];
				b_shared[tid] += b_shared[tid+16];
				x_shared[tid] += x_shared[tid+16];
				y_shared[tid] += y_shared[tid+16];
				xw_shared[tid] += xw_shared[tid+16];
				yw_shared[tid] += yw_shared[tid+16];
				zw_shared[tid] += zw_shared[tid+16];
				size_shared[tid] += size_shared[tid+16];
				num_of_points[tid] += num_of_points[tid+16];
			}
			if(blockSize >= 16){
				r_shared[tid] += r_shared[tid+8];
				g_shared[tid] += g_shared[tid+8];
				b_shared[tid] += b_shared[tid+8];
				x_shared[tid] += x_shared[tid+8];
				y_shared[tid] += y_shared[tid+8];
				xw_shared[tid] += xw_shared[tid+8];
				yw_shared[tid] += yw_shared[tid+8];
				zw_shared[tid] += zw_shared[tid+8];
				size_shared[tid] += size_shared[tid+8];
				num_of_points[tid] += num_of_points[tid+8];
			}
			if(blockSize >= 8){
				r_shared[tid] += r_shared[tid+4];
				g_shared[tid] += g_shared[tid+4];
				b_shared[tid] += b_shared[tid+4];
				x_shared[tid] += x_shared[tid+4];
				y_shared[tid] += y_shared[tid+4];
				xw_shared[tid] += xw_shared[tid+4];
				yw_shared[tid] += yw_shared[tid+4];
				zw_shared[tid] += zw_shared[tid+4];
				size_shared[tid] += size_shared[tid+4];
				num_of_points[tid] += num_of_points[tid+4];
			}
			if(blockSize >= 4){
				r_shared[tid] += r_shared[tid+2];
				g_shared[tid] += g_shared[tid+2];
				b_shared[tid] += b_shared[tid+2];
				x_shared[tid] += x_shared[tid+2];
				y_shared[tid] += y_shared[tid+2];
				xw_shared[tid] += xw_shared[tid+2];
				yw_shared[tid] += yw_shared[tid+2];
				zw_shared[tid] += zw_shared[tid+2];
				size_shared[tid] += size_shared[tid+2];
				num_of_points[tid] += num_of_points[tid+2];
			}
			if(blockSize >= 2){
				r_shared[tid] += r_shared[tid+1];
				g_shared[tid] += g_shared[tid+1];
				b_shared[tid] += b_shared[tid+1];
				x_shared[tid] += x_shared[tid+1];
				y_shared[tid] += y_shared[tid+1];
				xw_shared[tid] += xw_shared[tid+1];
				yw_shared[tid] += yw_shared[tid+1];
				zw_shared[tid] += zw_shared[tid+1];
				size_shared[tid] += size_shared[tid+1];
				num_of_points[tid] += num_of_points[tid+1];
			}
		}
		//store center point
		if(tid == 0){
			if(size_shared[0] != 0){
				int r = r_shared[0]/size_shared[0]>255 ? 255:r_shared[0]/size_shared[0];
				int g = g_shared[0]/size_shared[0]>255 ? 255:g_shared[0]/size_shared[0];
				int b = b_shared[0]/size_shared[0]>255 ? 255:b_shared[0]/size_shared[0];
				r = r<0 ? 0:r;
				g = g<0 ? 0:g;
				b = b<0 ? 0:b;
				int2 pixel;
				if(num_of_points[0] != 0){
					sp_centers[cluster_id].x = xw_shared[0]/(float)num_of_points[0];
					sp_centers[cluster_id].y = yw_shared[0]/(float)num_of_points[0];
					sp_centers[cluster_id].z = zw_shared[0]/(float)num_of_points[0];
					//real to projective
					float2 norm;
					norm.x = sp_centers[cluster_id].x/sp_centers[cluster_id].z;
					norm.y = sp_centers[cluster_id].y/sp_centers[cluster_id].z;
					pixel.x = (int)(norm.x*intr.data[0] + intr.data[2]);
					pixel.y = (int)(intr.data[5] - norm.y*intr.data[4]);
					if(pixel.x<0 || pixel.x>=width || pixel.y<0 || pixel.y<=height){
						pixel.x = x_shared[0]/size_shared[0];
						pixel.y = y_shared[0]/size_shared[0];
					}
				}
				else{
					pixel.x = x_shared[0]/size_shared[0];
					pixel.y = y_shared[0]/size_shared[0];
				}
				//pixel.x = pixel.x<0 ? 0:pixel.x;
				//pixel.x = pixel.x>=width ? width:pixel.x;
				//pixel.y = pixel.y<0 ? 0:pixel.y;
				//pixel.y = pixel.y>=height ? height:pixel.y;
				mean[cluster_id].x = pixel.x;
				mean[cluster_id].y = pixel.y;
				mean[cluster_id].r = (unsigned char)(r);
				mean[cluster_id].g = (unsigned char)(g);
				mean[cluster_id].b = (unsigned char)(b);
				mean[cluster_id].size = size_shared[0];
			}
		}

}

void DepthAdaptiveSuperpixel::Segmentation(cv::gpu::GpuMat color_image, float3* points3d_device, 
												float color_sigma, float spatial_sigma, float depth_sigma, int iteration){
		//init label distance
		init_LD<<<dim3(width/32, height/32), dim3(32, 32)>>>
			(LD_Device, width, height, ClusterNum, Window_Size);
		//sample clusters, move centers
		sampleInitialClusters<4*4><<<dim3(ClusterNum.x, ClusterNum.y), dim3(4, 4)>>>
			(color_image, points3d_device, meanData_Device, superpixelCenters_Device, width, height, Window_Size);
		for(int i = 0; i < iteration; i++){
			//Set cluster IDs	
			calculateLD<4*4><<<dim3(width, height), dim3(4, 4)>>>
				(color_image, points3d_device, LD_Device, meanData_Device, superpixelCenters_Device, Labels_Device, 
				Window_Size, width, height, color_sigma, spatial_sigma, depth_sigma, ClusterNum, Intrinsic_Device);
			analyzeClusters<16*16><<<dim3(ClusterNum.x, ClusterNum.y), dim3(16, 16)>>>
				(color_image, points3d_device, LD_Device, meanData_Device, superpixelCenters_Device, 
				Window_Size, ClusterNum, width, height, Intrinsic_Device);
		}
		hipMemcpy(Labels_Host, Labels_Device, sizeof(int)*width*height, hipMemcpyDeviceToHost);
}
