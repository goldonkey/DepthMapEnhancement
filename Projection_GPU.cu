#include "hip/hip_runtime.h"
#include "Projection_GPU.h"
#include "DimensionConvertor.h"

__global__ void initTemp(float3* temp, int width, int height){
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	temp[x + y * width].x = x;
	temp[x + y * width].y = y;
	temp[x + y * width].z = 1;
}

__global__ void setPsuedoDepth(
	const float3* input_3d, 
	float3* plane_fitted, 
	float3* normalized, 
	const float4* nd, 
	const int* labels, 
	const float* variance, 
	int width, int height){
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;


	int l = labels[x + y * width];
	//float a = nd[l].x;
	//float b = nd[l].y;
	//float c = nd[l].z;
	//float d = nd[l].w;
	if(l > -1 && acos(variance[l]) < (3.141592653f / 8.0f)){
		float a = nd[y*width+x].x;
		float b = nd[y*width+x].y;
		float c = nd[y*width+x].z;
		float d = nd[y*width+x].w;

		float3* ref = &plane_fitted[x + y * width];
		ref->z = abs(d / (a * normalized[x + y * width].x + b * normalized[x + y * width].y + c));
		ref->x = ref->z*normalized[x + y * width].x;
		ref->y = ref->z*normalized[x + y * width].y;
	}
	else{
		plane_fitted[x + y * width].x = input_3d[y*width+x].x;
		plane_fitted[x + y * width].y = input_3d[y*width+x].y;
		plane_fitted[x + y * width].z = input_3d[y*width+x].z;
	}
}
__global__ void setPsuedoDepth(
	const float3* input_3d, 
	float3* plane_fitted, 
	float3* normalized, 
	const float3* normals, 
	const float3* centers, 
	const int* labels, 
	const float* variance, 
	int width, int height){
		int x = blockIdx.x * blockDim.x + threadIdx.x;
		int y = blockIdx.y * blockDim.y + threadIdx.y;


		int l = labels[x + y * width];
		float a = normals[l].x;
		float b = normals[l].y;
		float c = normals[l].z;
		float d = fabs(a*centers[l].x+b*centers[l].y+c*centers[l].z);
		if(l > -1){
			//float a = nd[y*width+x].x;
			//float b = nd[y*width+x].y;
			//float c = nd[y*width+x].z;
			//float d = nd[y*width+x].w;
			if(acos(variance[l]) <  (3.141592653f / 8.0f)){
				float3* ref = &plane_fitted[x + y * width];
				ref->z = abs(d / (a * normalized[x + y * width].x + b * normalized[x + y * width].y + c));
				ref->x = ref->z*normalized[x + y * width].x;
				ref->y = ref->z*normalized[x + y * width].y;
			}
			else{
				plane_fitted[x + y * width].x = input_3d[y*width+x].x;
				plane_fitted[x + y * width].y = input_3d[y*width+x].y;
				plane_fitted[x + y * width].z = input_3d[y*width+x].z;
			}
		}
		else{
			plane_fitted[x + y * width].x = input_3d[y*width+x].x;
			plane_fitted[x + y * width].y = input_3d[y*width+x].y;
			plane_fitted[x + y * width].z = input_3d[y*width+x].z;
		}
}
//void Projection_GPU::getProjectedMap(){
//	//initialize
//	initTemp<<<dim3(width / 32, height / 24), dim3(32, 24)>>>
//		(Normalized3D_Device, width, height);
//	//prepare for projection
//	dim->projectiveToReal(Normalized3D_Device, Normalized3D_Device, width*height);
//	//plane projection
//	setPsuedoDepth<<<dim3(width / 32, height / 24), dim3(32, 24)>>>
//		(Upsampled3D_Device, Normalized3D_Device, normal_device, labels_device, width, height);
//	
//}

void Projection_GPU::initNormalized3D(){
	//initialize
	initTemp<<<dim3(width / 32, height / 24), dim3(32, 24)>>>
		(Normalized3D_Device, width, height);
	//prepare for projection
	dim->projectiveToReal(Normalized3D_Device, Normalized3D_Device);
}
__global__ void getFinalizedOutputKernel(const float3* input3d, float3* planefitted3d,
	//const float4* nd,
	float3* normalized_3d,
	const int* labels,
	int width,
	int height){
		int x = blockIdx.x * blockDim.x + threadIdx.x;
		int y = blockIdx.y * blockDim.y + threadIdx.y;
		int access = x + y * width;
		int l = labels[access];

		if(l == -1){
			//�_�̐��Ŕ��f
			planefitted3d[access].x = input3d[access].x;
			planefitted3d[access].y = input3d[access].y;
			planefitted3d[access].z = input3d[access].z;
		}

		//Interpolation�Ƃ̔�r
		else if(input3d[access].z > 50.0){
			float distance = sqrt(pow(planefitted3d[access].x-input3d[access].x, 2) +
				pow(planefitted3d[access].y-input3d[access].y, 2) +
				pow(planefitted3d[access].z-input3d[access].z, 2));
			//���ʂɂ���Ƃ��ꂪ�傫���_�i���ʂɂ͂ł��Ȃ��j
			if(distance >= 500.0){
				///���ςƂ��Ă�����
				/*planefitted3d[access].x = (planefitted3d[access].x+input3d[access].x)/2.0;
				planefitted3d[access].y = (planefitted3d[access].y+input3d[access].y)/2.0;
				planefitted3d[access].z = (planefitted3d[access].z+input3d[access].z)/2.0;*/
				planefitted3d[access].x = input3d[access].x;
				planefitted3d[access].y = input3d[access].y;
				planefitted3d[access].z = input3d[access].z;
			}
			else if(distance > 300.0){
				planefitted3d[access].x = (planefitted3d[access].x+input3d[access].x)/2.0;
				planefitted3d[access].y = (planefitted3d[access].y+input3d[access].y)/2.0;
				planefitted3d[access].z = (planefitted3d[access].z+input3d[access].z)/2.0;
				/*planefitted3d[access].z = ((7.0-distance)*planefitted3d[access].z+(distance-3.0)*input3d[access].z)/4.0;
				planefitted3d[access].x = ((7.0-distance)*planefitted3d[access].x+(distance-3.0)*input3d[access].x)/4.0;
				planefitted3d[access].y = ((7.0-distance)*planefitted3d[access].y+(distance-3.0)*input3d[access].y)/4.0;*/
			}
		}
		//Interpolation�̓_���Ȃ��Ƃ�
		else {
			//Occlusion���m�C�Y���𔻒f
			int count_neighber = 0; 
			//scan right
			float3 right_point;
			right_point.x = 0.0;
			right_point.y = 0.0;
			right_point.z = 0.0;
			int scan_right = 0;
			bool rightExist = false;
			while(x+scan_right < width && 
				labels[access+scan_right]==l &&
				rightExist==false){
					scan_right++;			
					if(input3d[access+scan_right].z > 50){
						right_point.x = input3d[access+scan_right].x;
						right_point.y = input3d[access+scan_right].y;
						right_point.z = input3d[access+scan_right].z;
						rightExist = true;
						count_neighber++;
					}
			};

			//scan left
			float3 left_point;
			int scan_left = 0;
			left_point.x = 0.0;
			left_point.y = 0.0;
			left_point.z = 0.0;
			bool leftExist = false;
			while(x-scan_left >= 0 && 
				labels[access-scan_left]==l &&
				leftExist == false){
					scan_left++;
					if(input3d[access-scan_left].z > 50){
						left_point.x = input3d[access-scan_left].x;
						left_point.y = input3d[access-scan_left].y;
						left_point.z = input3d[access-scan_left].z;
						leftExist = true;
						count_neighber++;
					}
					scan_left++;
			};
			//scan up
			float3 up_point;
			up_point.x = 0.0;
			up_point.y = 0.0;
			up_point.z = 0.0;
			int scan_up = 0;
			bool upExist = false;
			while(y-scan_up >= 0 && 
				labels[access-scan_up*width]==l &&
				upExist == false){
					scan_up++;
					if(input3d[access-scan_up*width].z > 50){
						up_point.x = input3d[access-scan_up*width].x;
						up_point.y = input3d[access-scan_up*width].y;
						up_point.z = input3d[access-scan_up*width].z;
						upExist = true;
						count_neighber++;
					}
			};
			//scan down
			float3 down_point;
			down_point.x = 0.0;
			down_point.y = 0.0;
			down_point.z = 0.0;
			int scan_down = 0;
			bool downExist = false;
			while(y+scan_down < height && 
				labels[access+scan_down*width]==l &&
				downExist == false){
					scan_down++;
					if(input3d[access+scan_down*width].z > 50){
						down_point.x = input3d[access+scan_down*width].x;
						down_point.y = input3d[access+scan_down*width].y;
						down_point.z = input3d[access+scan_down*width].z;
						downExist = true;
						count_neighber++;
					}
			};
			//Occulusion�̂Ƃ�
			if(count_neighber < 2){
				planefitted3d[access].x = 0.0;
				planefitted3d[access].y = 0.0;
				planefitted3d[access].z = 0.0;
			}
			//Occulusion�ł͂Ȃ��Ƃ�
			else{
				//interpolation���N���X�^���ł����Ȃ�
				//��������interpolation
				float3 interpolate_horizontal;
				if(rightExist && leftExist){
					interpolate_horizontal.x = (left_point.x*scan_right + right_point.x*scan_left)/(float)(scan_right+scan_left); 
					interpolate_horizontal.y = (left_point.y*scan_right + right_point.y*scan_left)/(float)(scan_right+scan_left);
					interpolate_horizontal.z = (left_point.z*scan_right + right_point.z*scan_left)/(float)(scan_right+scan_left);
				}
				else{
					interpolate_horizontal.x = left_point.x + right_point.x;
					interpolate_horizontal.y = left_point.y + right_point.y;
					interpolate_horizontal.z = left_point.z + right_point.z;
				}
				//�c������interpolation
				float3 interpolate_vertical;
				if(upExist && downExist){
					interpolate_vertical.x = (up_point.x*scan_down + down_point.x*scan_up)/(float)(scan_up+scan_down); 
					interpolate_vertical.y = (up_point.y*scan_down + down_point.y*scan_up)/(float)(scan_up+scan_down);
					interpolate_vertical.z = (up_point.z*scan_down + down_point.z*scan_up)/(float)(scan_up+scan_down);
				}
				else{
					interpolate_vertical.x = up_point.x + down_point.x;
					interpolate_vertical.y = up_point.y + down_point.y;
					interpolate_vertical.z = up_point.z + down_point.z;
				}
				//interpolation
				float3 interpolate;
				if(interpolate_horizontal.z > 50.0 && interpolate_vertical.z > 50.0){
					//interpolate.x = (interpolate_horizontal.x + interpolate_vertical.x) / 2.0;
					//interpolate.y = (interpolate_horizontal.y + interpolate_vertical.y) / 2.0;
					interpolate.z = (interpolate_horizontal.z + interpolate_vertical.z) / 2.0;
					interpolate.x = interpolate.z * normalized_3d[access].x;
					interpolate.y = interpolate.z * normalized_3d[access].y;
				}
				else{
					//interpolate.x = interpolate_horizontal.x + interpolate_vertical.x;
					//interpolate.y = interpolate_horizontal.y + interpolate_vertical.y;
					interpolate.z = interpolate_horizontal.z + interpolate_vertical.z;
					interpolate.x = interpolate.z * normalized_3d[access].x;
					interpolate.y = interpolate.z * normalized_3d[access].y; 
				}
				//���ʂ�interpolation�̍�������
				float distance = sqrt(pow(planefitted3d[access].x-interpolate.x, 2) +
					pow(planefitted3d[access].y-interpolate.y, 2) +
					pow(planefitted3d[access].z-interpolate.z, 2));
				if(distance > 500.0){
					if(count_neighber == 2){
						planefitted3d[access].x = 0.0;
						planefitted3d[access].y = 0.0;
						planefitted3d[access].z = 0.0;
					}
					else{
						planefitted3d[access].x = interpolate.x;
						planefitted3d[access].y = interpolate.y;
						planefitted3d[access].z = interpolate.z;
					}
					/*planefitted3d[access].x = 0.0;
					planefitted3d[access].y = 0.0;
					planefitted3d[access].z = 0.0;*/
				}
				else if(distance > 300.0){
					planefitted3d[access].x = interpolate.x;
					planefitted3d[access].y = interpolate.y;
					planefitted3d[access].z = interpolate.z;
				}
				else if(distance > 200.0){
					planefitted3d[access].x = (planefitted3d[access].x + interpolate.x)/2.0;
					planefitted3d[access].y = (planefitted3d[access].y + interpolate.y)/2.0;
					planefitted3d[access].z = (planefitted3d[access].z + interpolate.z)/2.0;
				}
			}
		}
}



__device__ void _atomicMin(double* address, double* val){
	double old = *address, assumed;
	do{
		assumed = old;
		old = 
			__longlong_as_double(
			atomicCAS(
			(unsigned long long int*)address, 
			__double_as_longlong(assumed), 
			__double_as_longlong(
			(*((float*)val) > *((float*)&assumed)) ? assumed : *val				
			)
			)
			);

	}while(assumed != old);
}


__global__ void makeJustifiedDepthMap(const float3* interpolate3D, const float2* interpolate_img, float3* depth_out, int width, int height){
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	depth_out[x + y * width].x=0.0;
	depth_out[x + y * width].y=0.0;
	depth_out[x + y * width].z=0.0;

	int2 j_coordinate;
	j_coordinate.x = (int)(interpolate_img[x + y * width].x+0.5);
	j_coordinate.y = (int)(interpolate_img[x + y * width].y+0.5);

	if(j_coordinate.x >= 0.0 && j_coordinate.y >= 0.0 &&
		j_coordinate.x < width && j_coordinate.y < height){

			//_atomicMin((double*)&depth_out[j_coordinate.x + j_coordinate.y * width], (double*)&interpolate3D[x + y * width].z);
			depth_out[j_coordinate.x + j_coordinate.y * width].x = interpolate3D[x + y * width].x;
			depth_out[j_coordinate.x + j_coordinate.y * width].y = interpolate3D[x + y * width].y;
			depth_out[j_coordinate.x + j_coordinate.y * width].z = interpolate3D[x + y * width].z;
	}
	//depth_out[j_coordinate.x + j_coordinate.y * width] = interpolate3D[x + y * width].z
}
__global__ void mrf_optimization(
	const float3* input3d,
	float3* planefitted3d,
	float3* normalized_3d,
	const int* labels,
	int width,
	int height,
	int window_size,
	float K,
	float smooth_sigma){
		int x = blockIdx.x * blockDim.x + threadIdx.x;
		int y = blockIdx.y * blockDim.y + threadIdx.y;

		if(planefitted3d[y*width+x].z > 50.0f){
		//mrf optimization
		float numerator = planefitted3d[y*width+x].z, denominator = 1.0f;
		for(int i = - window_size/2; i <= window_size/2; i++){		// y
			for(int j = -window_size/2; j <= window_size/2; j++){		// x
				int xj = x+j, yi = y+i;
				if(xj >= 0 && xj < width && yi >= 0 && yi < height && input3d[yi*width+xj].z > 50.0f ){
					//float distance = sqrt(pow(planefitted3d[y*width+x].x-input3d[y*width+x].x, 2) +
					//						pow(planefitted3d[y*width+x].y-input3d[y*width+x].y, 2) +
					//							pow(planefitted3d[y*width+x].z-input3d[y*width+x].z, 2));
					float diff = fabs(input3d[y*width+x].z-input3d[yi*width+xj].z);
					float depth_filter = K/(1+pow(diff, 2.0f));
					//calculate filter
					float filter = smooth_sigma*depth_filter;
					numerator += input3d[yi*width+xj].z*filter; 
					denominator += filter;
				}
			}
		}
		if(denominator != 0.0f){
			float depth = numerator/denominator;	
			planefitted3d[y*width+x].z = depth;
			planefitted3d[y*width+x].x = normalized_3d[y*width+x].x*depth;
			planefitted3d[y*width+x].y = normalized_3d[y*width+x].y*depth;
			}
		}
}
__global__ void variance_optimization(
	const float3* input3d,
	const float* variance, 
	float3* planefitted3d,
	float3* normalized_3d,
	const int* labels,
	int width,
	int height){
		int x = blockIdx.x * blockDim.x + threadIdx.x;
		int y = blockIdx.y * blockDim.y + threadIdx.y;

		if(planefitted3d[y*width+x].z > 50.0f){
				if(fabs((float)planefitted3d[y*width+x].z-(float)input3d[y*width+x].z)>input3d[y*width+x].z*0.01f){
						planefitted3d[y*width+x].x = input3d[y*width+x].x;
						planefitted3d[y*width+x].y = input3d[y*width+x].y;
						planefitted3d[y*width+x].z = input3d[y*width+x].z;
				}
				else if(labels[y*width+x] > -1 && (acos(variance[labels[y*width+x]]) < (3.141592653f / 8.0f))){
						planefitted3d[y*width+x].z = planefitted3d[y*width+x].z*variance[labels[y*width+x]]+input3d[y*width+x].z*(1.0f-variance[labels[y*width+x]]);
						//planefitted3d[y*width+x].z = planefitted3d[y*width+x].z*(1.0f-variance[y*width+x])+input3d[y*width+x].z*variance[y*width+x];
						planefitted3d[y*width+x].x = normalized_3d[y*width+x].x*planefitted3d[y*width+x].z;
						planefitted3d[y*width+x].y = normalized_3d[y*width+x].y*planefitted3d[y*width+x].z;
				}
		
		}
}
void Projection_GPU::PlaneProjection(const float4* nd_device, const int* labels_device, const float* variance_device, const float3* points3d_device){
	//���ׂĂ̓_�𕽖�fitting����
	//getProjectedMap();
	//plane projection
	setPsuedoDepth<<<dim3(width / 32, height / 24), dim3(32, 24)>>>
		(points3d_device, PlaneFitted3D_Device, Normalized3D_Device, nd_device, labels_device, variance_device, width, height);

	//Input�Ƃ̔�r
	//getFinalizedOutputKernel<<<dim3(width / 32, height / 24), dim3(32, 24)>>>
	//	(points3d_device, PlaneFitted3D_Device, Normalized3D_Device, labels_device, width, height);
	//mrf_optimization<<<dim3(width / 32, height / 24), dim3(32, 24)>>>
	//		(points3d_device, PlaneFitted3D_Device, Normalized3D_Device, labels_device, width, height, 5, 0.5f, 1.0f);
	variance_optimization<<<dim3(width / 32, height / 24), dim3(32, 24)>>>
			(points3d_device, variance_device, PlaneFitted3D_Device, Normalized3D_Device, labels_device, width, height);

	//3D��2D
	//dim->realToProjective2(PlaneFitted3D_Device, Upsampled2D_Device, width*height);
	//Device to Host
	hipMemcpy(PlaneFitted3D_Host, PlaneFitted3D_Device, width * height * sizeof(float3), hipMemcpyDeviceToHost);
	//for(int y=0; y<height; y++){
	//	for(int x=0; x<width; x++){
	//		std::cout << PlaneFitted3D_Host[y*width+x].z <<std::endl;
	//	}
	//}
}

void Projection_GPU::PlaneProjection(
	const float3* normals_device, 
	const float3* centers_device, 
	const int* labels_device,
	const float* variance_device, 
	const float3* points3d_device){	
		//���ׂĂ̓_�𕽖�fitting����
		//getProjectedMap();
		//plane projection
		setPsuedoDepth<<<dim3(width / 32, height / 24), dim3(32, 24)>>>
			(points3d_device, PlaneFitted3D_Device, Normalized3D_Device, 
			normals_device, centers_device, labels_device, variance_device, width, height);

		//Input�Ƃ̔�r
		//getFinalizedOutputKernel<<<dim3(width / 32, height / 24), dim3(32, 24)>>>
		//	(points3d_device, PlaneFitted3D_Device, Normalized3D_Device, labels_device, width, height);
		mrf_optimization<<<dim3(width / 32, height / 24), dim3(32, 24)>>>
			(points3d_device, PlaneFitted3D_Device, Normalized3D_Device, labels_device, width, height, 15, 200.0f, 200.0f);

		//3D��2D
		//dim->realToProjective2(PlaneFitted3D_Device, Upsampled2D_Device, width*height);
		//Device to Host
		hipMemcpy(PlaneFitted3D_Host, PlaneFitted3D_Device, width * height * sizeof(float3), hipMemcpyDeviceToHost);
		//for(int y=0; y<height; y++){
		//	for(int x=0; x<width; x++){
		//		std::cout << PlaneFitted3D_Host[y*width+x].z <<std::endl;
		//	}
		//}
}